#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime_api.h>
#include "device2.hpp"


template <typename LOOP_BODY>
__global__ void forall_kernel_gpu2(int start, int length, LOOP_BODY body)
{
  int idx = blockDim.x * blockIdx.x + threadIdx.x;

  if (idx < length) {
    body(idx);
  }
}


template <typename LOOP_BODY>
void forall(camp::devices::Cuda dev, int begin, int end, LOOP_BODY&& body)
{
  size_t blockSize = 32;
  size_t gridSize = (end - begin + blockSize - 1) / blockSize;

  forall_kernel_gpu2<<<gridSize, blockSize, 0, dev.get_stream()>>>(begin, end - begin, body);
}


// This is a kernel that does no real work but runs at least for a specified number of clocks
__global__ void clock_block_a(clock_t clock_count)
{
  unsigned int start_clock = (unsigned int) clock();
  clock_t clock_offset = 0;
  while (clock_offset < clock_count)
  {
    unsigned int end_clock = (unsigned int) clock();
    clock_offset = (clock_t)(end_clock - start_clock);
  }
}


int main(int argc, char *argv[])
{
  float kernel_time = 20; // time the kernel should run in ms
  int cuda_device = 0;
  int N = 30000;
  
  hipDeviceProp_t deviceProp;
  hipGetDevice(&cuda_device);
  hipGetDeviceProperties(&deviceProp, cuda_device);
  if ((deviceProp.concurrentKernels == 0))
  {
    printf("> GPU does not support concurrent kernel execution\n");
    printf("  CUDA kernel runs will be serialized\n");
  }
  printf("> Detected Compute SM %d.%d hardware with %d multi-processors\n",
   deviceProp.major, deviceProp.minor, deviceProp.multiProcessorCount);

#if defined(__arm__) || defined(__aarch64__)
  clock_t time_clocks = (clock_t)(kernel_time * (deviceProp.clockRate / 1000));
#else
  clock_t time_clocks = (clock_t)(kernel_time * deviceProp.clockRate);
#endif


  // -----------------------------------------------------------------------

  camp::devices::Cuda cudev1;
  camp::devices::Cuda cudev2;
  float * m1 = cudev1.allocate<float>(N);
  float * m2 = cudev2.allocate<float>(N);


  auto clock_lambda_1 = [=] __device__ (int idx) {
    m1[idx] = idx * 2;
    unsigned int start_clock = (unsigned int) clock();
    clock_t clock_offset = 0;
    while (clock_offset < time_clocks)
    {
      unsigned int end_clock = (unsigned int) clock();
      clock_offset = (clock_t)(end_clock - start_clock);
    }
  };

  auto clock_lambda_2 = [=] __device__ (int idx) {
    m2[idx] = 1234;
    unsigned int start_clock = (unsigned int) clock();
    clock_t clock_offset = 0;
    while (clock_offset < time_clocks)
    {
      unsigned int end_clock = (unsigned int) clock();
      clock_offset = (clock_t)(end_clock - start_clock);
    }
  };

  auto clock_lambda_3 = [=] __device__ (int idx) {
    float val = m1[idx];
    m1[idx] = val * val;
    unsigned int start_clock = (unsigned int) clock();
    clock_t clock_offset = 0;
    while (clock_offset < time_clocks)
    {
      unsigned int end_clock = (unsigned int) clock();
      clock_offset = (clock_t)(end_clock - start_clock);
    }
  };


  forall(cudev1, 0, N, clock_lambda_1);
  forall(cudev2, 0, N, clock_lambda_2);
  forall(cudev1, 0, N, clock_lambda_3);

  hipDeviceSynchronize();

  // -----------------------------------------------------------------------
  

  std::cout << "---------- M1 = (idx * 2) ^ 2 ----------" << std::endl;
  for (int i = 0; i < 15; i++) {
    std::cout << m1[i] << std::endl;
  }

  std::cout << "---------- M2 = 1234 ----------" << std::endl;
  for (int i = 0; i < 15; i++) {
    std::cout << m2[i] << std::endl;
  }

  hipDeviceReset();
  return 0;
}
