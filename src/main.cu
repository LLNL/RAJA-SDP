#include "hip/hip_runtime.h"
#include <iostream>
#include "forall.hpp"

#define HOST_DEVICE __host__ __device__

using namespace std;

int main(int argc, char *argv[])
{
  int option = 0;
  if (argc > 1) option = atoi(argv[1]);  

  auto lambda = [] HOST_DEVICE (int tid) {
    int sum = 0;
    for (int i = 0; i < 10000; i++){
      sum += sqrt(pow(3.14159,i));
    }
  };

  
  switch(option){ 
    case 0:
      std::cout << "Running Sequentially" << std::endl;
      sequential s;
      forall(s, 0, 30000, lambda);
      break;
    case 1:
      std::cout << "Running On GPU" << std::endl;
      gpu g; 
      auto dev = camp::devices::CudaDevice::get(0);
      forall(g, dev, 0, 30000, lambda);
      break;
  }

  return 0;
}
